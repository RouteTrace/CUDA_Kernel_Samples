#include "hip/hip_runtime.h"
#include <stdio.h>
#include "utils.cuh"

void _cudaCheck(hipError_t error, const char *file, int line) {
    if (error != hipSuccess) {
        printf("[CUDA ERROR] at file %s(line %d):\n%s\n", file, line, hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
    return;
};

void CudaDeviceInfo() {
    int deviceId;

    hipGetDevice(&deviceId);

    hipDeviceProp_t props;
    hipGetDeviceProperties(&props, deviceId);

    /*
   * There should be no need to modify the output string below.
   */

    printf("Device ID: %d\n\
       *Number of SMs: %d\n\
       Compute Capability Major: %d\n\
       Compute Capability Minor: %d\n\
       memoryBusWidth: %d\n\
       *maxThreadsPerBlock: %d\n\
       maxThreadsPerMultiProcessor: %d\n\
       *totalGlobalMem: %zuM\n\
       sharedMemPerBlock: %zuKB\n\
       *sharedMemPerMultiprocessor: %zuKB\n\
       totalConstMem: %zuKB\n\
       *multiProcessorCount: %d\n\
       *Warp Size: %d\n",
           deviceId,
           props.multiProcessorCount,
           props.major,
           props.minor,
           props.memoryBusWidth,
           props.maxThreadsPerBlock,
           props.maxThreadsPerMultiProcessor,
           props.totalGlobalMem / 1024 / 1024,
           props.sharedMemPerBlock / 1024,
           props.sharedMemPerMultiprocessor / 1024,
           props.totalConstMem / 1024,
           props.multiProcessorCount,
           props.warpSize);
};

void randomize_matrix(float *mat, size_t N) {
    // NOTICE: 使用gettimeofdays替代srand((unsigned)time(NULL));time精度过低，产生相同随机数
    struct timeval time;
    gettimeofday(&time, NULL);
    srand(time.tv_usec);
    for (int i = 0; i < N; i++) {
        float tmp = (float) (rand() % 5) + 0.01 * (rand() % 5);
        tmp = (rand() % 2 == 0) ? tmp : tmp * (-1.);
        mat[i] = tmp;
    }
}

void copy_matrix(float *src, float *dest, size_t N) {
    int i;
    for (i = 0; src + i && dest + i && i < N; i++)
        *(dest + i) = *(src + i);
    if (i != N)
        printf("copy failed at %d while there are %lu elements in total.\n", i, N);
}

void print_matrix(const float *A, int M, int N) {
    int i;
    printf("[");
    for (i = 0; i < M * N; i++) {
        if ((i + 1) % N == 0)
            printf("%5.2f ", A[i]);
        else
            printf("%5.2f, ", A[i]);
        if ((i + 1) % N == 0) {
            if (i + 1 < M * N)
                printf(";\n");
        }
    }
    printf("]\n");
}

bool verify_matrix(float *mat1, float *mat2, size_t N) {
    double diff = 0.0;
    int i;
    for (i = 0; mat1 + i && mat2 + i && i < N; i++) {
        diff = fabs((double) mat1[i] - (double) mat2[i]);
        if (diff > 1e-2) {
            printf("error. %5.2f,%5.2f,%d\n", mat1[i], mat2[i], i);
            return false;
        }
    }
    return true;
}

float call_kernel(int kernel_num, bool record, int M, int N, int K, float alpha, float *A, float *B, float beta, float *C) {
    float total_time = 0;
    float repeat_times = 0;
    if (record) repeat_times = REPEAT_TIMES;
    if (kernel_num == 0) {
        hipblasHandle_t handle;
        hipblasCreate(&handle);
        total_time = TIME_RECORD(repeat_times, ([&]{hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, B, N, A, K, &beta, C, N);}));
        hipblasDestroy(handle);
    }
    else if (kernel_num == 1) {
        dim3 blockDim(32, 32);
        dim3 gridDim(CEIL_DIV(M, 32), CEIL_DIV(N, 32));
        total_time = TIME_RECORD(repeat_times, ([&]{sgemm_v1<<<gridDim, blockDim>>>(M, N, K, alpha, A, B, beta, C);}));
        
    } else {
        printf("Error: kernel %d not found.\n", kernel_num);
        exit(EXIT_FAILURE);
    }
    return total_time;
}