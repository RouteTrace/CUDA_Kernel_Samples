#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "utils.cuh"

void host_transpose(float* input, int M, int N, float* output) {
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < M; j++) {
            output[i * M + j] = input[j * N + i];
        }
    }
}

// 朴素实现
__global__ void device_transpose_v0(const float* input, float* output, int M, int N) {
    const int row = blockDim.y * blockIdx.y + threadIdx.y;
    const int col = blockDim.x * blockIdx.x + threadIdx.x;

    if (row < M && col < N) {
        output[col * M + row] = input[row * N + col];
    }
}

// 合并写入，减少不合并读取的影响
__global__ void device_transpose_v1(const float* input, float* output, int M, int N) {
    const int row = blockDim.y * blockIdx.y + threadIdx.y;
    const int col = blockDim.x * blockIdx.x + threadIdx.x;

    if (row < N && col < M) {
        output[row * M + col] = input[col * N + row];
    }
}

// 显式调用__ldg，减少不合并读取的影响
__global__ void device_transpose_v2(const float* input, float* output, int M, int N) {
    const int row = blockDim.y * blockIdx.y + threadIdx.y;
    const int col = blockDim.x * blockIdx.x + threadIdx.x;

    if (row < N && col < M) {
        output[row * M + col] = __ldg(&input[col * N + row]);
    }
}

// 使用共享内存中转，合并读取+写入
template <const int TILE_DIM>
__global__ void device_transpose_v3(const float* input, float* output, int M, int N) {
    __shared__ float S[TILE_DIM][TILE_DIM];
    const int bx = blockDim.x * TILE_DIM;
    const int by = blockDim.y * TILE_DIM;
    const int x1 = bx + threadIdx.x;
    const int y1 = by + threadIdx.y;

    if (y1 < M && x1 < N) {
        S[threadIdx.y][threadIdx.x] = input[y1 * N + x1];  // 合并读取
    }
    __syncthreads();

    const int x2 = by + threadIdx.x;
    const int y2 = bx + threadIdx.y;
    if (y2 < N && x2 < M) {
        output[y2 * M + x2] = S[threadIdx.y][threadIdx.x];  // 合并写入
    }
}

int main() {
    size_t M = 12800;
    size_t N = 12800;
    constexpr size_t BLOCK_SIZE = 32;
    const int repeat_times = 10;

    // 1. host
    float *h_matrix = (float *)malloc(sizeof(float) * M * N);
    float *h_matrix_tr = (float *)malloc(sizeof(float) * N * M);
    randomize_matrix(h_matrix, M * N);
    host_transpose(h_matrix, M, N, h_matrix_tr);
    // printf("init_matrix:\n");
    // print_matrix(h_matrix, M, N);
    // printf("host_transpose:\n");
    // print_matrix(h_matrix_tr, N, M);

    // 2. device
    float *d_matrix, *d_matrix_tr;
    hipMalloc((void **) &d_matrix, sizeof(float) * M * N);
    hipMalloc((void **) &d_matrix_tr, sizeof(float) * M * N);
    hipMemcpy(d_matrix, h_matrix, sizeof(float) * M * N, hipMemcpyHostToDevice);

    // 2.1 call transpose_v0
    dim3 block_size0(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid_size0(CEIL(M, BLOCK_SIZE), CEIL(N, BLOCK_SIZE));
    float total_time0 = TIME_RECORD(repeat_times, ([&]{device_transpose_v0<<<grid_size0, block_size0>>>(d_matrix, d_matrix_tr, M, N);}));
    hipMemcpy(h_matrix_tr, d_matrix_tr, sizeof(float) * M * N, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    printf("[device_transpose_v0] Average time: (%f) seconds\n", total_time0 / repeat_times);
    // print_matrix(h_matrix_tr, N, M);
    
    // 2.2 call transpose_v1
    dim3 block_size1(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid_size1(CEIL(N, BLOCK_SIZE), CEIL(M, BLOCK_SIZE));
    float total_time1 = TIME_RECORD(repeat_times, ([&]{device_transpose_v1<<<grid_size1, block_size1>>>(d_matrix, d_matrix_tr, M, N);}));
    hipMemcpy(h_matrix_tr, d_matrix_tr, sizeof(float) * M * N, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    printf("[device_transpose_v1] Average time: (%f) seconds\n", total_time1 / repeat_times);
    // print_matrix(h_matrix_tr, N, M);

    // 2.3 call transpose_v2
    dim3 block_size2(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid_size2(CEIL(N, BLOCK_SIZE), CEIL(M, BLOCK_SIZE));
    float total_time2 = TIME_RECORD(repeat_times, ([&]{device_transpose_v2<<<grid_size2, block_size2>>>(d_matrix, d_matrix_tr, M, N);}));
    hipMemcpy(h_matrix_tr, d_matrix_tr, sizeof(float) * M * N, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    printf("[device_transpose_v2] Average time: (%f) seconds\n", total_time2 / repeat_times);
    // print_matrix(h_matrix_tr, N, M);

    // 2.4 call transpose_v3
    dim3 block_size3(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid_size3(CEIL(N, BLOCK_SIZE) * CEIL(M, BLOCK_SIZE));
    float total_time3 = TIME_RECORD(repeat_times, ([&]{device_transpose_v3<BLOCK_SIZE><<<grid_size3, block_size3>>>(d_matrix, d_matrix_tr, M, N);}));
    hipMemcpy(h_matrix_tr, d_matrix_tr, sizeof(float) * M * N, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    printf("[device_transpose_v3] Average time: (%f) seconds\n", total_time3 / repeat_times);
    // print_matrix(h_matrix_tr, N, M);

    // free memory
    free(h_matrix);
    free(h_matrix_tr);
    hipFree(d_matrix);
    hipFree(d_matrix_tr);
    return 0;
}